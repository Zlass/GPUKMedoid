
#include <hip/hip_runtime.h>

// Number of threads per block.
#define NT 1024

// Structure for a 3-D point.
typedef struct {
  double x;
  double y;
  double z;
}point_t;

// Structure for a solution.
typedef struct {
  int a;
  int b;
  double d;
}solution_t;

// Variables in global memory.
__device__ int devBestSol;

// Per-thread variables in shared memory.
__shared__ solution_t shrSols[NT];
__shared__ int shrBestSolIndex[NT];

/**
 *  Calculates the city-block distance between two point_t structs as defiend
 *  by this function.  distance(P1,P2) = |x1 − x2| + |y1 − y2| + |z1 − z2|
 *  @param    p1    A pointer to the first point.
 *  @param    p2    A pointer to the second point.
 *  @return   The city block distance between p1 and p2.
 */
__device__ double distance(point_t *p1, point_t *p2) {

  double tempX = p1->x - p2->x;
  if (tempX < 0) {
    tempX *= -1;
  }
  double tempY = p1->y - p2->y;
  if (tempY < 0) {
    tempY *= -1;
  }
  double tempZ = p1->z - p2->z;
  if (tempZ < 0) {
    tempZ *= -1;
  }
  return tempX + tempY + tempZ;
}

/**
 * Compares to different solution_t to find the one with the lowest distance then a index and finall b index.
 * @param a   Pointer to first solution.
 * @param b   Pointer to second solution.
 * @return  true if a is the better solution false otherwise.
 */
__device__ bool compareSol(solution_t *a, solution_t *b){
  bool aIsbest = false;
  if(a->d == -1.0){aIsbest = false;}
  else if(b->d == -1.0){aIsbest = true;}
  else if (b->d > a->d) {
    aIsbest = true;
  } else if (b->d == a->d) {
    if (b->a > a->a){
      aIsbest = true;
    }else{
      if (b->b > a->b){
        aIsbest = true;
      }
    }
  }
  return aIsbest;
}

/**
 * Device kernel to calculate the distance for each point to its closest medoid.
 *
 * Called with a one-dimensional grid of one-dimensional blocks, N blocks, NT
 * threads per block. N = number of points. Each block finds the best solution
 * for its given A index. Each thread within a block computes its total
 * distance for its B index(s).
 *
 * @param  pointList   Array of all the points.
 * @param  solutions   Array of all the solutions the gpu finds.
 * @param  N           Total number of points.
 */
extern "C" __global__ void computeMedoids
    (point_t *pointList, solution_t *solutions, int N) {
  int a = blockIdx.x, b; // X index of this block, the A medoids index
  int thrd = threadIdx.x; // Index of this thread within block
  point_t medA = pointList[a]; // Medoid A's point
  // double *solu = &solutions[a + b * N];
  double d = 0.0;
  solution_t sol;
  sol.a = a;
  sol.b = thrd;
  sol.d = -1.0;

  for (b = thrd; b < N; b += NT) {
    d = 0;
    solution_t tempSol;
    for (int p = 0; p < N; p++){
      point_t medB = pointList[b]; // Medoid B's point
      if (p == a || p == b) continue;
      double distA = distance( &medA, &pointList[p]);
      double distB = distance( &medB, &pointList[p]);
      d += (distA <= distB) ? distA : distB;
      tempSol.a = a;
      tempSol.b = b;
      tempSol.d = (b <= a | b >= N) ? -1 : d;
    }
    if(!compareSol(&sol, &tempSol))
      sol = tempSol;
  }
  shrSols[thrd] = sol;
  shrBestSolIndex[thrd] = thrd;
  __syncthreads();

  // Reduction to find the best solution in the current block
  for (int s = NT / 2; s > 0; s >>= 1) {
    if (thrd < s){
      shrBestSolIndex[thrd] = compareSol(&shrSols[shrBestSolIndex[thrd]], &shrSols[shrBestSolIndex[thrd + s]]) ? shrBestSolIndex[thrd] : shrBestSolIndex[thrd + s];

    }
    __syncthreads();
  }
  solutions[a] = shrSols[shrBestSolIndex[0]];

  __threadfence();

  //  Final reduction to find the best solution overall.
  if (thrd == 0) {
    int oldSol;
    int newSol;
    do {
      oldSol = devBestSol;
      newSol = compareSol(&solutions[devBestSol], &solutions[a]) ? devBestSol : a;
    } while (atomicCAS (&devBestSol, oldSol, newSol) != oldSol);
  }
}
